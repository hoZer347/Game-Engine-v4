#include "hip/hip_runtime.h"
#include "Cuda.cuh"

#include "Matrix.h"

#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <iostream>

#include <chrono>
#include <thread>

namespace loom
{
	__global__
	void matrixMultiplyKernel(const float* m0, const float* m1, float* m2, const uint32_t rows, const uint32_t cols, const uint32_t commonDim)
	{
		// Calculate the row and column indices for the current thread
		uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
		uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;

		// Perform matrix multiplication for the current element
		if (row < rows && col < cols)
		{
			float value = 0.0f;
			for (uint32_t k = 0; k < commonDim; ++k)
			{
				value += m0[row * commonDim + k] * m1[k * cols + col];
			}
			m2[row * cols + col] = value;
		}
	};



	void Mat2D::multiply(Mat2D& m0, Mat2D& m1, Mat2D& m2)
	{
		// Check if matrix dimensions are compatible for multiplication
		if (m0.cols != m1.rows || m0.rows != m2.rows || m1.cols != m2.cols)
			throw std::invalid_argument("Incompatible matrix dimensions for multiplication");

		// Allocate GPU memory for matrices
		float* d_m0, * d_m1, * d_m2;
		size_t sizeM0 = m0.rows * m0.cols * sizeof(float);
		size_t sizeM1 = m1.rows * m1.cols * sizeof(float);
		size_t sizeM2 = m2.rows * m2.cols * sizeof(float);
		hipMalloc((void**)&d_m0, sizeM0);
		hipMalloc((void**)&d_m1, sizeM1);
		hipMalloc((void**)&d_m2, sizeM2);

		// Copy matrix data from host to GPU memory
		hipMemcpy(d_m0, m0.data, sizeM0, hipMemcpyHostToDevice);
		hipMemcpy(d_m1, m1.data, sizeM1, hipMemcpyHostToDevice);

		// Set the CUDA kernel launch configuration
		dim3 blockSize(16, 16);  // 16x16 thread block
		dim3 gridSize((m2.cols + blockSize.x - 1) / blockSize.x, (m2.rows + blockSize.y - 1) / blockSize.y);  // grid size based on output matrix dimensions

		// Launch the matrix multiplication kernel on the GPU
		matrixMultiplyKernel<<<gridSize, blockSize>>>(d_m0, d_m1, d_m2, m2.rows, m2.cols, m0.cols);

		// Copy the result matrix from GPU to host memory
		hipMemcpy(m2.data, d_m2, sizeM2, hipMemcpyDeviceToHost);

		// Free GPU memory
		hipFree(d_m0);
		hipFree(d_m1);
		hipFree(d_m2);
	};
};
